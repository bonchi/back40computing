#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a scan of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for scan.
 ******************************************************************************/

#include <stdio.h> 
#include <b40c/scan/enactor.cuh>

// Test utils
#include "b40c_test_util.h"


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Struct for doing addition and max scan simultaneously
 */
struct Foo
{
	int a, b;

	__host__ __device__ __forceinline__ Foo() :
		a(0), b(0) {}

	__host__ __device__ __forceinline__ Foo(int a, int b) :
		a(a), b(b) {}

	__host__ __device__ __forceinline__ bool operator == (const Foo& other) const
	{
		return ((a == other.a) && (b == other.b));
	}

	__host__ __device__ __forceinline__ bool operator != (const Foo& other) const
	{
		return ((a != other.a) || (b != other.b));
	}

	void Print()
	{
		printf("[a: %d, b: %d]", a, b);
	}
};


/**
 * Foo binary scan operator
 */
struct MultiScan
{
	// Associative reduction operator
	__host__ __device__ __forceinline__ Foo operator()(const Foo &x, const Foo &y)
	{
		return Foo(
			x.a + y.a,
			(x.b > y.b) ? x.b : y.b);
	}

	// Identity operator
	__host__ __device__ __forceinline__ Foo operator()()
	{
		return Foo();
	}

	enum {
		NON_COMMUTATIVE = true,
	};
};


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	b40c::CommandLineArgs args(argc, argv);

	// Usage/help
    if (args.CheckCmdLineFlag("help") || args.CheckCmdLineFlag("h")) {
    	printf("\nsimple_scan [--device=<device index>]\n");
    	return 0;
    }

    b40c::DeviceInit(args);
    int num_elements = 564;
    bool verbose = args.CheckCmdLineFlag("v");
    bool exclusive = args.CheckCmdLineFlag("exclusive");
    args.GetCmdLineArgument("n", num_elements);

	// Allocate and initialize host problem data and host reference solution
	Foo *h_src = new Foo[num_elements];
	Foo *h_reference = new Foo[num_elements];
	MultiScan max_op;

	for (size_t i = 0; i < num_elements; ++i) {
		h_src[i] = Foo(i, i);

		if (exclusive) {
			h_reference[i] = (i == 0) ?
				max_op() :									// identity
				max_op(h_reference[i - 1], h_src[i - 1]);
		} else {
			h_reference[i] = (i == 0) ?
				h_src[i] :
				max_op(h_reference[i - 1], h_src[i]);
		}
	}

	
	// Allocate and initialize device data
	Foo *d_src, *d_dest;
	hipMalloc((void**) &d_src, sizeof(Foo) * num_elements);
	hipMalloc((void**) &d_dest, sizeof(Foo) * num_elements);
	hipMemcpy(d_src, h_src, sizeof(Foo) * num_elements, hipMemcpyHostToDevice);


	// Create a scan enactor
	b40c::scan::Enactor scan_enactor;

	// Enact simple exclusive scan using internal tuning heuristics
	if (exclusive) {
		scan_enactor.Scan<true, MultiScan::NON_COMMUTATIVE>(
			d_dest, d_src, num_elements, max_op, max_op);
	} else {
		scan_enactor.Scan<false, MultiScan::NON_COMMUTATIVE>(
			d_dest, d_src, num_elements, max_op, max_op);
	}
	
	printf("Simple scan: "); b40c::CompareDeviceResults(h_reference, d_dest, num_elements, verbose, verbose); printf("\n");

	delete h_src;
	delete h_reference;

	return 0;
}

