/******************************************************************************
 * 
 * Reductionright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a reduction of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for reduction.
 ******************************************************************************/

#include <stdio.h> 
#include <b40c/reduction/enactor.cuh>

// Test utils
#include "b40c_test_util.h"

using namespace b40c;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/


/**
 * Max binary associative operator
 */
template <typename T>
__host__ __device__ __forceinline__ T Max(const T &a, const T &b)
{
	return (a > b) ? a : b;
}


/**
 * Example showing syntax for invoking templated member functions from 
 * a templated function
 */
template <
	typename T,
	T BinaryOp(const T&, const T&)>
void TemplatedSubroutineReduction(
	b40c::reduction::Enactor &reduction_enactor,
	T *d_dest, 
	T *d_src,
	int num_elements)
{
	reduction_enactor.template Reduce<T, BinaryOp>(d_dest, d_src, num_elements);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	// Usage/help
    if (args.CheckCmdLineFlag("help") || args.CheckCmdLineFlag("h")) {
    	printf("\nsimple_reduction [--device=<device index>]\n");
    	return 0;
    }

    DeviceInit(args);

	typedef unsigned int T;
	const int NUM_ELEMENTS = 10;

	// Allocate and initialize host problem data and host reference solution
	T h_data[NUM_ELEMENTS];
	T h_reference[1];
	for (int i = 0; i < NUM_ELEMENTS; i++) {
		h_data[i] = i;
		h_reference[0] = (i == 0) ?
			h_data[i] :
			Max(h_reference[0], h_data[i]);
	}
	
	// Allocate and initialize device data
	T *d_src, *d_dest;
	hipMalloc((void**) &d_src, sizeof(T) * NUM_ELEMENTS);
	hipMalloc((void**) &d_dest, sizeof(T) * NUM_ELEMENTS);
	hipMemcpy(d_src, h_data, sizeof(T) * NUM_ELEMENTS, hipMemcpyHostToDevice);
	
	// Create a reduction enactor
	b40c::reduction::Enactor reduction_enactor;
	

	//
	// Example 1: Enact simple reduction using internal tuning heuristics
	//
	reduction_enactor.Reduce<T, Max>(d_dest, d_src, NUM_ELEMENTS);
	
	printf("Simple reduction: "); CompareDeviceResults(h_reference, d_dest, 1); printf("\n");
	
	
	//
	// Example 2: Enact simple reduction using "large problem" tuning configuration
	//
	reduction_enactor.Reduce<T, Max, b40c::reduction::LARGE_SIZE>(
		d_dest, d_src, NUM_ELEMENTS);

	printf("Large-tuned reduction: "); CompareDeviceResults(h_reference, d_dest, 1); printf("\n");

	
	//
	// Example 3: Enact simple reduction using "small problem" tuning configuration
	//
	reduction_enactor.Reduce<T, Max, b40c::reduction::SMALL_SIZE>(
		d_dest, d_src, NUM_ELEMENTS);
	
	printf("Small-tuned reduction: "); CompareDeviceResults(h_reference, d_dest, 1); printf("\n");


	//
	// Example 4: Enact simple reduction using a templated subroutine function
	//
	TemplatedSubroutineReduction<T, Max>(reduction_enactor, d_dest, d_src, NUM_ELEMENTS);
	
	printf("Templated subroutine reduction: "); CompareDeviceResults(h_reference, d_dest, 1); printf("\n");


	//
	// Example 5: Enact simple reduction using custom tuning configuration (base reduction enactor)
	//

	typedef b40c::reduction::ProblemType<T, size_t, Max> ProblemType;
	typedef b40c::reduction::Policy<
		ProblemType,
		b40c::reduction::SM20,
		b40c::util::io::ld::cg,
		b40c::util::io::st::cg,
		true,
		false,
		true, 
		false, 
		8, 7, 1, 2, 9,
		8, 1, 1> CustomPolicy;
	
	reduction_enactor.Reduce<CustomPolicy>(d_dest, d_src, NUM_ELEMENTS);

	printf("Custom reduction: "); CompareDeviceResults(h_reference, d_dest, 1); printf("\n");

	return 0;
}

