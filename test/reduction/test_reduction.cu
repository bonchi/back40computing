#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for reduction.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_reduction.h"

using namespace b40c;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool 	g_verbose 						= false;
bool 	g_sweep							= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;



/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntest_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Creates an example reduction problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename SizeT,
	typename ReductionOp>
void TestReduction(
	SizeT num_elements,
	ReductionOp reduction_op)
{
    // Allocate the reduction problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(sizeof(T));

	if ((h_data == NULL) || (h_reference == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
		// util::RandomBits<T>(h_data[i], 0);
		h_data[i] = i;
		h_reference[0] = (i == 0) ?
			h_data[i] :
			reduction_op(h_reference[0], h_data[i]);
	}

	//
    // Run the timing test(s)
	//


	// Execute test(s), optionally sweeping problem size downward
	size_t orig_num_elements = num_elements;
	do {
		printf("\nLARGE config:\t");
		double large = TimedReduction<reduction::LARGE_SIZE>(
			h_data, h_reference, num_elements, reduction_op, g_max_ctas, g_verbose, g_iterations);

		printf("\nSMALL config:\t");
		double small = TimedReduction<reduction::SMALL_SIZE>(
			h_data, h_reference, num_elements, reduction_op, g_max_ctas, g_verbose, g_iterations);

		if (small > large) {
			printf("%lu-byte elements: Small faster at %lu elements\n", (unsigned long) sizeof(T), (unsigned long) num_elements);
		}

		num_elements -= 4096;

	} while (g_sweep && (num_elements < orig_num_elements ));

	// Free our allocated host memory
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    g_sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 4, reduction_op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements * 2, reduction_op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements, reduction_op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> reduction_op;
    	TestReduction<T>(num_elements / 2, reduction_op);
	}

	return 0;
}



