#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Tuning tool for establishing optimal reduction granularity configuration types
 ******************************************************************************/

#include <stdio.h> 

// Reduction includes
#include <b40c/reduction/problem_type.cuh>
#include <b40c/reduction/policy.cuh>
#include <b40c/reduction/enactor.cuh>
#include <b40c/util/arch_dispatch.cuh>
#include <b40c/util/cuda_properties.cuh>
#include <b40c/util/numeric_traits.cuh>
#include <b40c/util/parameter_generation.cuh>

// Test utils
#include "b40c_test_util.h"

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals, and utility types
 ******************************************************************************/

#ifndef TUNE_ARCH
	#define TUNE_ARCH (200)
#endif

bool g_verbose;
int g_max_ctas = 0;
int g_iterations = 0;


template <typename T>
struct Sum
{
	static __host__ __device__ __forceinline__ T BinaryOp(const T &a, const T &b)
	{
		return a + b;
	}
};

template <typename T>
struct Max
{
	static __host__ __device__ __forceinline__ T BinaryOp(const T &a, const T &b)
	{
		return (a > b) ? a : b;
	}
};



/******************************************************************************
 * Utility routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntune_reduction [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>]\n");
	printf("\n");
	printf("\t--v\tDisplays verbose configuration to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the reduction operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}


/**
 * Enumerated tuning params
 */
enum TuningParam {

	PARAM_BEGIN,

		WORK_STEALING,

		UNIFORM_SMEM_ALLOCATION,
		UNIFORM_GRID_SIZE,
		OVERSUBSCRIBED_GRID_SIZE,

		UPSWEEP_LOG_THREADS,
		UPSWEEP_LOG_LOAD_VEC_SIZE,
		UPSWEEP_LOG_LOADS_PER_TILE,

	PARAM_END,

	// Parameters below here are currently not part of the tuning sweep
	READ_MODIFIER,
	WRITE_MODIFIER,
	UPSWEEP_MAX_CTA_OCCUPANCY,

	// Derive these from the others above
	LOG_SCHEDULE_GRANULARITY,

	// General performance is insensitive to the spine kernel params
	// because it's only a single-CTA: we'll just use reasonable defaults
	SPINE_LOG_THREADS,
	SPINE_LOG_LOAD_VEC_SIZE,
	SPINE_LOG_LOADS_PER_TILE,
};



/**
 * Encapsulation structure for
 * 		- Wrapping problem type and storage
 * 		- Providing call-back for parameter-list generation
 */
template <typename T, typename OpType>
class TuneEnactor : public reduction::Enactor
{
public:

	T *d_dest;
	T *d_src;
	T *h_data;
	T *h_reference;
	size_t num_elements;

	/**
	 * Ranges for the tuning params
	 */
	template <typename ParamList, int PARAM> struct Ranges;

	// READ_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, READ_MODIFIER> {
		enum {
			MIN = util::io::ld::NONE,
			MAX = ((TUNE_ARCH < 200) || (util::NumericTraits<T>::REPRESENTATION == util::NOT_A_NUMBER)) ? util::io::ld::NONE : util::io::ld::LIMIT - 1		// No type modifiers for pre-Fermi or non-builtin types
		};
	};

	// WRITE_MODIFIER
	template <typename ParamList>
	struct Ranges<ParamList, WRITE_MODIFIER> {
		enum {
			MIN = util::io::st::NONE,
			MAX = ((TUNE_ARCH < 200) || (util::NumericTraits<T>::REPRESENTATION == util::NOT_A_NUMBER)) ? util::io::st::NONE : util::io::st::LIMIT - 1		// No type modifiers for pre-Fermi or non-builtin types
		};
	};

	// UNIFORM_SMEM_ALLOCATION
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_SMEM_ALLOCATION> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};

	// UNIFORM_GRID_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, UNIFORM_GRID_SIZE> {
		enum {
			MIN = 0,
			MAX = 1
		};
	};

	// OVERSUBSCRIBED_GRID_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, OVERSUBSCRIBED_GRID_SIZE> {
		enum {
			MIN = 0,
			MAX = !util::Access<ParamList, WORK_STEALING>::VALUE		// Don't oversubscribe if we're workstealing
		};
	};

	// WORK_STEALING
	template <typename ParamList>
	struct Ranges<ParamList, WORK_STEALING> {
		enum {
			MIN = 0,
			MAX = (TUNE_ARCH < 200) ? 0 : 1				// Only bother tuning atomic worstealing on Fermi+
		};
	};

	// UPSWEEP_LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, UPSWEEP_LOG_THREADS> {
		enum {
			MIN = B40C_LOG_WARP_THREADS(TUNE_ARCH),
			MAX = B40C_LOG_CTA_THREADS(TUNE_ARCH)
		};
	};

	// UPSWEEP_LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, UPSWEEP_LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// UPSWEEP_LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, UPSWEEP_LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// SPINE_LOG_THREADS
	template <typename ParamList>
	struct Ranges<ParamList, SPINE_LOG_THREADS> {
		enum {
			MIN = B40C_LOG_WARP_THREADS(TUNE_ARCH),
			MAX = B40C_LOG_CTA_THREADS(TUNE_ARCH)
		};
	};

	// SPINE_LOG_LOAD_VEC_SIZE
	template <typename ParamList>
	struct Ranges<ParamList, SPINE_LOG_LOAD_VEC_SIZE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	// SPINE_LOG_LOADS_PER_TILE
	template <typename ParamList>
	struct Ranges<ParamList, SPINE_LOG_LOADS_PER_TILE> {
		enum {
			MIN = 0,
			MAX = 2
		};
	};

	/**
	 * Constructor
	 */
	TuneEnactor(size_t num_elements) :
		reduction::Enactor(), d_dest(NULL), d_src(NULL), h_data(NULL), h_reference(NULL), num_elements(num_elements) {}


	/**
	 * Timed scan for applying a specific granularity configuration type
	 */
	template <typename Policy>
	void TimedReduction()
	{
		printf("%lu, ", (unsigned long) sizeof(T));
		Policy::Print();
		fflush(stdout);

		// Perform a single iteration to allocate any memory if needed, prime code caches, etc.
		this->ENACTOR_DEBUG = g_verbose;
		if (this->template Reduce<Policy>(d_dest, d_src, num_elements, g_max_ctas)) {
			exit(1);
		}
		this->ENACTOR_DEBUG = false;

		// Perform the timed number of iterations

		hipEvent_t start_event, stop_event;
		hipEventCreate(&start_event);
		hipEventCreate(&stop_event);

		double elapsed = 0;
		float duration = 0;
		for (int i = 0; i < g_iterations; i++) {

			// Start cuda timing record
			hipEventRecord(start_event, 0);

			// Call the scan API routine
			if (this->template Reduce<Policy>(d_dest, d_src, num_elements, g_max_ctas)) {
				exit(1);
			}

			// End cuda timing record
			hipEventRecord(stop_event, 0);
			hipEventSynchronize(stop_event);
			hipEventElapsedTime(&duration, start_event, stop_event);
			elapsed += (double) duration;

			// Flushes any stdio from the GPU
			hipDeviceSynchronize();
		}

		// Display timing information
		double avg_runtime = elapsed / g_iterations;
		double throughput =  0.0;
		if (avg_runtime > 0.0) throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
	    printf(", %f, %f, %f, ",
			avg_runtime, throughput, throughput * sizeof(T));
	    fflush(stdout);

	    // Clean up events
		hipEventDestroy(start_event);
		hipEventDestroy(stop_event);

	    // Copy out data
	    if (util::B40CPerror(hipMemcpy(h_data, d_dest, sizeof(T) * 1, hipMemcpyDeviceToHost),
			"TimedReduction hipMemcpy d_dest failed: ", __FILE__, __LINE__)) exit(1);

	    // Verify solution
		CompareResults<T>(h_data, h_reference, 1, true);
		printf("\n");
		fflush(stdout);
	}


	/**
	 * Callback invoked by parameter-list generation
	 */
	template <typename ParamList>
	void Invoke()
	{
		const int C_READ_MODIFIER =
//			util::Access<ParamList, READ_MODIFIER>::VALUE;
			util::io::ld::NONE;
		const int C_WRITE_MODIFIER =
//			util::Access<ParamList, WRITE_MODIFIER>::VALUE;
			util::io::st::NONE;
		const int C_UNIFORM_SMEM_ALLOCATION =
			util::Access<ParamList, UNIFORM_SMEM_ALLOCATION>::VALUE;
//			0;
		const int C_UNIFORM_GRID_SIZE =
			util::Access<ParamList, UNIFORM_GRID_SIZE>::VALUE;
//			0;
		const int C_OVERSUBSCRIBED_GRID_SIZE =
			util::Access<ParamList, OVERSUBSCRIBED_GRID_SIZE>::VALUE;
//			0;

		const int C_WORK_STEALING =
			util::Access<ParamList, WORK_STEALING>::VALUE;
//			0;
		const int C_UPSWEEP_LOG_THREADS =
			util::Access<ParamList, UPSWEEP_LOG_THREADS>::VALUE;
//			5;//
		const int C_UPSWEEP_LOG_LOAD_VEC_SIZE =
			util::Access<ParamList, UPSWEEP_LOG_LOAD_VEC_SIZE>::VALUE;
//			0;
		const int C_UPSWEEP_LOG_LOADS_PER_TILE =
			util::Access<ParamList, UPSWEEP_LOG_LOADS_PER_TILE>::VALUE;
//			0;
		const int C_UPSWEEP_MAX_CTA_OCCUPANCY =
//			util::Access<ParamList, UPSWEEP_MAX_CTA_OCCUPANCY>::VALUE;
			B40C_SM_CTAS(TUNE_ARCH);

		const int C_UPSWEEP_LOG_SCHEDULE_GRANULARITY =
			C_UPSWEEP_LOG_LOADS_PER_TILE +
			C_UPSWEEP_LOG_LOAD_VEC_SIZE +
			C_UPSWEEP_LOG_THREADS;

		// General performance is insensitive to spine config it's only a single-CTA:
		// simply use reasonable defaults
		const int C_SPINE_LOG_THREADS =
//			util::Access<ParamList, SPINE_LOG_THREADS>::VALUE;
			8;
		const int C_SPINE_LOG_LOAD_VEC_SIZE =
//			util::Access<ParamList, SPINE_LOG_LOAD_VEC_SIZE>::VALUE;
			0;
		const int C_SPINE_LOG_LOADS_PER_TILE =
//			util::Access<ParamList, SPINE_LOG_LOADS_PER_TILE>::VALUE;
			1;

		// Establish the problem type
		typedef reduction::ProblemType<
			T,
			size_t,
			OpType::BinaryOp> ProblemType;

		// Establish the granularity configuration type
		typedef reduction::Policy <
			ProblemType,
			TUNE_ARCH,
			(util::io::ld::CacheModifier) C_READ_MODIFIER,
			(util::io::st::CacheModifier) C_WRITE_MODIFIER,
			C_WORK_STEALING,
			C_UNIFORM_SMEM_ALLOCATION,
			C_UNIFORM_GRID_SIZE,
			C_OVERSUBSCRIBED_GRID_SIZE,

			C_UPSWEEP_MAX_CTA_OCCUPANCY,
			C_UPSWEEP_LOG_THREADS,
			C_UPSWEEP_LOG_LOAD_VEC_SIZE,
			C_UPSWEEP_LOG_LOADS_PER_TILE,
			C_UPSWEEP_LOG_SCHEDULE_GRANULARITY,

			C_SPINE_LOG_THREADS,
			C_SPINE_LOG_LOAD_VEC_SIZE,
			C_SPINE_LOG_LOADS_PER_TILE> Policy;

		// Invoke this config
		TimedReduction<Policy>();
	}
};


/**
 * Creates an example scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<typename T, typename OpType>
void TestReduction(size_t num_elements)
{
	// Allocate storage and enactor
	typedef TuneEnactor<T, OpType> Detail;
	Detail detail(num_elements);

	if (util::B40CPerror(hipMalloc((void**) &detail.d_src, sizeof(T) * num_elements),
		"TimedReduction hipMalloc d_src failed: ", __FILE__, __LINE__)) exit(1);

	if (util::B40CPerror(hipMalloc((void**) &detail.d_dest, sizeof(T) * 1),
		"TimedReduction hipMalloc d_dest failed: ", __FILE__, __LINE__)) exit(1);

	if ((detail.h_data = (T*) malloc(sizeof(T) * num_elements)) == NULL) {
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}
	if ((detail.h_reference = (T*) malloc(sizeof(T) * 1)) == NULL) {
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
		// util::RandomBits<T>(detail.h_data[i], 0);
		detail.h_data[i] = i;
		detail.h_reference[0] = (i == 0) ?
			detail.h_data[i] :
			OpType::BinaryOp(detail.h_reference[0], detail.h_data[i]);
	}

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(detail.d_src, detail.h_data, sizeof(T) * num_elements, hipMemcpyHostToDevice),
		"TimedReduction hipMemcpy d_src failed: ", __FILE__, __LINE__)) exit(1);

	// Run the timing tests
	util::ParamListSweep<
		Detail,
		PARAM_BEGIN + 1,
		PARAM_END,
		Detail::template Ranges>::template Invoke<util::EmptyTuple>(detail);

	// Free allocated memory
	if (detail.d_src) hipFree(detail.d_src);
	if (detail.d_dest) hipFree(detail.d_dest);

	// Free our allocated host memory
	if (detail.h_data) free(detail.h_data);
	if (detail.h_reference) free(detail.h_reference);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	util::CudaProperties cuda_props;

	printf("Test Reduction: %d iterations, %lu elements", g_iterations, (unsigned long) num_elements);
	printf("\nCodeGen: \t[device_sm_version: %d, kernel_ptx_version: %d]\n\n",
		cuda_props.device_sm_version, cuda_props.kernel_ptx_version);

	printf("sizeof(T), READ_MODIFIER, WRITE_MODIFIER, WORK_STEALING, UNIFORM_SMEM_ALLOCATION, UNIFORM_GRID_SIZE, OVERSUBSCRIBED_GRID_SIZE, "
		"UPSWEEP_MAX_CTA_OCCUPANCY, UPSWEEP_LOG_THREADS, UPSWEEP_LOG_LOAD_VEC_SIZE, UPSWEEP_LOG_LOADS_PER_TILE, UPSWEEP_LOG_SCHEDULE_GRANULARITY, "
		"SPINE_LOG_THREADS, SPINE_LOG_LOAD_VEC_SIZE, SPINE_LOG_LOADS_PER_TILE, "
		"elapsed time (ms), throughput (10^9 items/s), bandwidth (10^9 B/s), Correctness\n");

	// Execute test(s)
	{
		typedef unsigned char T;
		TestReduction<T, Sum<T> >(num_elements * 4);
	}
/*
	{
		typedef unsigned short T;
		TestReduction<T, Sum<T> >(num_elements * 2);
	}
	{
		typedef unsigned int T;
		TestReduction<T, Sum<T> >(num_elements);
	}
	{
		typedef unsigned long long T;
		TestReduction<T, Sum<T> >(num_elements / 2);
	}
*/
	return 0;
}

