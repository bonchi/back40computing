#include "hip/hip_runtime.h"
/******************************************************************************
 * 
 * Copyright 2010-2011 Duane Merrill
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for segmented scan.
 ******************************************************************************/

#include <stdio.h> 

// Test utils
#include "b40c_test_util.h"
#include "test_segmented_scan.h"

using namespace b40c;

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

bool 	g_verbose 						= false;
bool 	g_sweep							= false;
int 	g_max_ctas 						= 0;
int 	g_iterations  					= 1;
bool 	g_inclusive						= false;


/******************************************************************************
 * Utility Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage()
{
	printf("\ntest_segmented_scan [--device=<device index>] [--v] [--i=<num-iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--inclusive] [--sweep]\n");
	printf("\n");
	printf("\t--v\tDisplays copied results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the segmented scan operation <num-iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
}



/**
 * Creates an example segmented scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename Flag,
	bool EXCLUSIVE,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
void TestSegmentedScan(
	SizeT num_elements,
	ReductionOp scan_op,
	IdentityOp identity_op)
{
    // Allocate the segmented scan problem on the host and fill the keys with random bytes

	T *h_data 			= (T*) malloc(num_elements * sizeof(T));
	T *h_reference 		= (T*) malloc(num_elements * sizeof(T));
	Flag *h_flag_data	= (Flag*) malloc(num_elements * sizeof(Flag));

	if ((h_data == NULL) || (h_reference == NULL) || (h_flag_data == NULL)){
		fprintf(stderr, "Host malloc of problem data failed\n");
		exit(1);
	}

	for (size_t i = 0; i < num_elements; ++i) {
	}

	if (g_verbose) printf("Input problem: \n");
	for (int i = 0; i < num_elements; i++) {
/*
		h_data[i] = 1;
		h_flag_data[i] = (i % 11) == 0;
*/
		util::RandomBits<T>(h_data[i], 0);
		util::RandomBits<Flag>(h_flag_data[i], 2, 1);

		if (g_verbose) {
			printf("(%lld, %lld), ", (long long) h_data[i], (long long) h_flag_data[i]);
		}
	}
	if (g_verbose) printf("\n");


	for (size_t i = 0; i < num_elements; ++i) {
		if (EXCLUSIVE)
		{
			h_reference[i] = ((i == 0) || (h_flag_data[i])) ?
				identity_op() :
				scan_op(h_reference[i - 1], h_data[i - 1]);
		} else {
			h_reference[i] = ((i == 0) || (h_flag_data[i])) ?
				h_data[i] :
				scan_op(h_reference[i - 1], h_data[i]);
		}
	}

	//
    // Run the timing test(s)
	//

	// Execute test(s), optionally sweeping problem size downward
	size_t orig_num_elements = num_elements;
	do {

		printf("\nLARGE config:\t");
		double large = TimedSegmentedScan<EXCLUSIVE, segmented_scan::LARGE_SIZE>(
			h_data,
			h_flag_data,
			h_reference,
			num_elements,
			scan_op,
			identity_op,
			g_max_ctas,
			g_verbose,
			g_iterations);

		printf("\nSMALL config:\t");
		double small = TimedSegmentedScan<EXCLUSIVE, segmented_scan::SMALL_SIZE>(
			h_data,
			h_flag_data,
			h_reference,
			num_elements,
			scan_op,
			identity_op,
			g_max_ctas,
			g_verbose,
			g_iterations);

		if (small > large) {
			printf("%lu-byte elements: Small faster at %lu elements\n",
				(unsigned long) sizeof(T), (unsigned long) num_elements);
		}

		num_elements -= 4096;

	} while (g_sweep && (num_elements < orig_num_elements ));

	// Free our allocated host memory
	if (h_flag_data) free(h_flag_data);
	if (h_data) free(h_data);
    if (h_reference) free(h_reference);
}


/**
 * Creates an example segmented scan problem and then dispatches the problem
 * to the GPU for the given number of iterations, displaying runtime information.
 */
template<
	typename T,
	typename Flag,
	typename SizeT,
	typename ReductionOp,
	typename IdentityOp>
void TestSegmentedScanVariety(
	SizeT num_elements,
	ReductionOp scan_op,
	IdentityOp identity_op)
{
	if (g_inclusive) {
		TestSegmentedScan<T, Flag, false>(num_elements, scan_op, identity_op);
	} else {
		TestSegmentedScan<T, Flag, true>(num_elements, scan_op, identity_op);
	}
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}
    g_inclusive = args.CheckCmdLineFlag("inclusive");
    g_sweep = args.CheckCmdLineFlag("sweep");
    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
	g_verbose = args.CheckCmdLineFlag("v");

	typedef unsigned char Flag;


	{
		printf("\n-- UNSIGNED CHAR ----------------------------------------------\n");
		typedef unsigned char T;
		Sum<T> op;
		TestSegmentedScanVariety<T, Flag>(num_elements * 4, op, op);
	}
	{
		printf("\n-- UNSIGNED SHORT ----------------------------------------------\n");
		typedef unsigned short T;
		Sum<T> op;
		TestSegmentedScanVariety<T, Flag>(num_elements * 2, op, op);
	}
	{
		printf("\n-- UNSIGNED INT -----------------------------------------------\n");
		typedef unsigned int T;
		Sum<T> op;
		TestSegmentedScanVariety<T, Flag>(num_elements, op, op);
	}
	{
		printf("\n-- UNSIGNED LONG LONG -----------------------------------------\n");
		typedef unsigned long long T;
		Sum<T> op;
		TestSegmentedScanVariety<T, Flag>(num_elements / 2, op, op);
	}

	return 0;
}



