#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * Duane Merrill
 * 4/30/12
 *
 *
 *****************************************************************************/

#include <stdio.h>
#include <algorithm>

#include <thrust/device_func.h>
#include <thrust/gather.h>
#include <thrust/sort.h>

#include "b40c_test_util.h"


/******************************************************************************
 * Misc. utilities
 *****************************************************************************/

typedef unsigned int uint;

/**
 * Mark's pointer wrapper
 */
namespace my_dev
{
	template <typename T>
	struct dev_mem
	{
		T *raw;

		dev_mem(T *raw) : raw(raw) {}

		T* raw_p()
		{
			return raw;
		}
	};
} // namespace my_dev


/**
 * 96-bit uint4 comparator
 */
bool Uint4Compare96(uint4 elem1, uint4 elem2)
{
	if (elem1.x != elem2.x) {
		return (elem1.x < elem2.x);

	} else if (elem1.y != elem2.y) {
		return (elem1.y < elem2.y);

	} else {
		return (elem1.z < elem2.z);
	}
}


/******************************************************************************
 * 96-bit sorting code
 *****************************************************************************/

namespace original
{

	/**
	 * Extract 32-bit word from uint4
	 */
	template<int keyIdx>
	struct ExtractBits: public thrust::unary_function<uint4, uint>
	{
		__host__ __device__ __forceinline__ uint operator()(uint4 key) const
		{
			if (keyIdx == 0)
				return key.x;
			else if (keyIdx == 1)
				return key.y;
			else
				return key.z;
		}
	};


	/**
	 * Update permutation
	 */
	template<
		int keyIdx,
		typename KeyPtr,
		typename PermutationPtr,
		typename ExtractedPtr>
	void update_permutation(
		KeyPtr& keys,
		PermutationPtr& permutation,
		ExtractedPtr& temp, int N)
	{
		// permute the keys with the current reordering
		thrust::gather(
			permutation,
			permutation + N,
			thrust::make_transform_iterator(
				keys,
				ExtractBits<keyIdx> ()),
			temp);

		// stable_sort the permuted keys and update the permutation
		thrust::stable_sort_by_key(
			temp,
			temp + N,
			permutation);
	}


	/**
	 * Apply permutation
	 */
	template<
		typename KeyPtr,
		typename PermutationPtr,
		typename OutputPtr>
	void apply_permutation(
		KeyPtr& keys,
		PermutationPtr& permutation,
		OutputPtr& out,
		int N)
	{
		// permute the keys into out vector
		thrust::gather(
			permutation,
			permutation + N,
			keys,
			out);
	}


	/**
	 * Sort the lower 96-bits of a uint4 structure
	 */
	void thrust_sort_96b(
		my_dev::dev_mem<uint4> srcKeys,
		my_dev::dev_mem<uint4> sortedKeys,
		my_dev::dev_mem<uint> temp_buffer,
		my_dev::dev_mem<uint> permutation_buffer,
		int N)
	{

		// wrap raw pointer with a device_func
		thrust::device_func<uint4> keys = thrust::device_pointer_cast(
				srcKeys.raw_p());
		thrust::device_func<uint4> outKeys = thrust::device_pointer_cast(
				sortedKeys.raw_p());
		thrust::device_func<uint> temp = thrust::device_pointer_cast(
				temp_buffer.raw_p());
		thrust::device_func<uint> permutation = thrust::device_pointer_cast(
				permutation_buffer.raw_p());

		// initialize permutation to [0, 1, 2, ... ,N-1]
		thrust::sequence(permutation, permutation + N);

		// sort z, y, x
		// careful: note 2, 1, 0 key word order, NOT 0, 1, 2.
		update_permutation<2> (keys, permutation, temp, N);
		update_permutation<1> (keys, permutation, temp, N);
		update_permutation<0> (keys, permutation, temp, N);

		// Note: keys have not been modified
		// Note: permutation now maps unsorted keys to sorted order

		thrust::gather(permutation, permutation + N, keys, outKeys);
	}

} // namespace original



/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_elements 	= 450 * 1000;	// 450K 96-bit keys
	int device_id 		= 0;

	// Get device id from command line
	if (argc > 1) {
		device_id = atoi(argv[1]);
	}
	hipSetDevice(device_id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device_id);
	printf("Using device %d: %s\n", device_id, deviceProp.name);

    // Allocate and initialize 96-bit keys on host
	printf("Allocating...\n"); fflush(stdout);
    uint4 *h_keys = new uint4[num_elements];
    for (int i(0); i < num_elements; ++i) {

    	b40c::util::RandomBits(h_keys[i].x);
    	b40c::util::RandomBits(h_keys[i].y);
    	b40c::util::RandomBits(h_keys[i].z);
    }

    // Compute answer (sorted keys) on host
	printf("Computing reference answer...\n"); fflush(stdout);
    uint4 *h_reference_keys = new uint4[num_elements];
    memcpy(h_reference_keys, h_keys, sizeof(uint4) * num_elements);
	std::stable_sort(h_reference_keys, h_reference_keys + num_elements, Uint4Compare96);

    // Allocate keys on device
	printf("Allocating problem to GPU...\n"); fflush(stdout);
    uint4 *d_keys;
    hipMalloc((void**)&d_keys, sizeof(uint4) * num_elements);

	// Allocate sorted keys on device
    uint4 *d_sorted_keys;
	hipMalloc((void**)&d_sorted_keys, sizeof(uint4) * num_elements);

    // Allocate output permutation vector on device
	uint *d_permutation;
	hipMalloc((void**)&d_permutation, sizeof(uint) * num_elements);

	// Allocate temp buffers on device
	uint *d_temp0;
	uint *d_temp1;
	hipMalloc((void**)&d_temp0, sizeof(uint) * num_elements);
	hipMalloc((void**)&d_temp1, sizeof(uint) * num_elements);


	//
    // Thrust
	//

	// Copy problem to GPU
	printf("Thrust: copying problem to GPU...\n"); fflush(stdout);
	hipMemcpy(d_keys, h_keys, sizeof(uint4) * num_elements, hipMemcpyHostToDevice);

	// Thrust sort
	original::thrust_sort_96b(
		my_dev::dev_mem<uint4>(d_keys),
		my_dev::dev_mem<uint4>(d_sorted_keys),
		my_dev::dev_mem<uint>(d_temp1),
		my_dev::dev_mem<uint>(d_permutation),
		num_elements);

	// Copy out results and check answer
    uint4 *h_sorted_keys = new uint4[num_elements];
	hipMemcpy(h_sorted_keys, d_sorted_keys, sizeof(uint4) * num_elements, hipMemcpyDeviceToHost);
	bool correct = true;
	for (int i(0); i < num_elements; ++i) {

		if ((h_sorted_keys[i].z != h_reference_keys[i].z) ||
			(h_sorted_keys[i].y != h_reference_keys[i].y) ||
			(h_sorted_keys[i].x != h_reference_keys[i].x))
		{
			printf("Incorrect: [%d]: (%d,%d,%d) != (%d,%d,%d)\n",
				i,
				h_sorted_keys[i].z,
				h_sorted_keys[i].y,
				h_sorted_keys[i].x,
				h_reference_keys[i].z,
				h_reference_keys[i].y,
				h_reference_keys[i].x);

			correct = false;
			break;
		}
	}
	if (correct) {
		printf("Correct\n");
	}

    // Cleanup
    delete h_keys;
    delete h_sorted_keys;
    delete h_reference_keys;

    hipFree(d_keys);
    hipFree(d_sorted_keys);
    hipFree(d_permutation);
    hipFree(d_temp0);
    hipFree(d_temp1);

    return 0;

}
